#include "hip/hip_runtime.h"
#include "TIPL/tipl.hpp"
#include "TIPL/cuda/mem.hpp"
#include "TIPL/cuda/basic_image.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
__global__ void cuda_test(){
    ;
}

bool has_cuda = true;
int gpu_count = 0;
void distribute_gpu(void)
{
    static int cur_gpu = 0;
    static std::mutex m;
    std::lock_guard<std::mutex> lock(m);
    if(gpu_count <= 1)
        return;
    if(hipSetDevice(cur_gpu) != hipSuccess)
        tipl::out() << "hipSetDevice error:" << hipSetDevice(cur_gpu) << std::endl;
    ++cur_gpu;
    if(cur_gpu >= gpu_count)
        cur_gpu = 0;
}

std::vector<std::string> gpu_names;
void check_cuda(std::string& error_msg)
{
    tipl::progress p("Initiating CUDA");
    int Ver;
    if(hipGetDeviceCount(&gpu_count) != hipSuccess ||
       hipDriverGetVersion(&Ver) != hipSuccess)
    {
        error_msg = "cannot obtain GPU driver and device information (CUDA ERROR:";
        error_msg += std::to_string(int(hipGetDeviceCount(&gpu_count)));
        error_msg += "). Please update the Nvidia driver and install CUDA Toolkit.";
        return;
    }
    tipl::out() << "CUDA Driver Version: " << Ver << " CUDA Run Time Version: " << CUDART_VERSION << std::endl;
    cuda_test<<<1,1>>>();
    if(hipPeekAtLastError() != hipSuccess)
    {
        error_msg = "Failed to lauch cuda kernel:";
        error_msg += hipGetErrorName(hipGetLastError());
        error_msg += ". Please update Nvidia driver.";
        return;
    }

    tipl::out() << "Device Count:" << gpu_count << std::endl;
    for (int i = 0; i < gpu_count; i++)
    {
        tipl::out() << "Device Number:" << std::to_string(i) << std::endl;
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) != hipSuccess)
        {
            error_msg = "Cannot obtain device information. Please update Nvidia driver";
            return;
        }
        auto arch = prop.major*10+prop.minor;
        tipl::out() << "Arch: " << arch << std::endl;
        tipl::out() << "Device name: " << prop.name << std::endl;
        tipl::out() << "Memory Size (GB): " << float(prop.totalGlobalMem >> 20)/1024.0f << std::endl;
        tipl::out() << "Memory Clock Rate (KHz): " << prop.memoryClockRate << std::endl;
        tipl::out() << "Memory Bus Width (bits): " << prop.memoryBusWidth << std::endl;
        tipl::out() << "Peak Memory Bandwidth (GB/s): " << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << std::endl;
        gpu_names.push_back(prop.name);
    }
    has_cuda = true;
}

size_t linear_cuda(const tipl::image<3,float>& from,
                              tipl::vector<3> from_vs,
                              const tipl::image<3,float>& to,
                              tipl::vector<3> to_vs,
                              tipl::affine_transform<float>& arg,
                              tipl::reg::reg_type reg_type,
                              bool& terminated,
                              const float* bound)
{
    distribute_gpu();
    return tipl::reg::linear_mr<tipl::reg::mutual_information_cuda>
            (from,from_vs,to,to_vs,arg,reg_type,[&](void){return terminated;},
                0.01,bound != tipl::reg::narrow_bound,bound);
}


