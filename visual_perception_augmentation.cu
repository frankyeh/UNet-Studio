#include "hip/hip_runtime.h"
#include "TIPL/tipl.hpp"
#include "TIPL/cuda/mem.hpp"
#include "TIPL/cuda/basic_image.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template<typename T1,typename T2>
__global__ void cropping_at_kernel(T1 image,T2 label,tipl::vector<3> pos,float radius,float cropping_value)
{
    TIPL_FOR(index,image.size())
    {
        tipl::vector<3> dir(tipl::pixel_index<3>(index,image.shape()));
        dir -= pos;
        if(dir[0] > radius || dir[1] > radius || dir[2] > radius)
            return;
        auto length = dir.length();
        if(length > radius)
            return;
        if(label[index])
        {
            image[index] = cropping_value;
            label[index] = 0;
        }
    }
}
template<typename image_type,typename label_type,typename vector_type>
inline void cropping_at(image_type& image,label_type& label,const vector_type& pos,float radius,float cropping_value)
{
    TIPL_RUN(cropping_at_kernel,image.size())
            (tipl::make_shared(image),tipl::make_shared(label),pos,radius,cropping_value);
}

template<typename T>
__global__ void truncate_top_kernel(T from,size_t slices)
{
    TIPL_FOR(index,slices*from.plane_size())
    {
        from[from.size()-1-index] = 0;
    }
}
template<typename T>
inline void truncate_top(T& from,size_t slices)
{
    TIPL_RUN(truncate_top_kernel,slices*from.plane_size())
            (tipl::make_shared(from),slices);
}

template<typename T>
__global__ void truncate_bottom_kernel(T from,size_t slices)
{
    TIPL_FOR(index,slices*from.plane_size())
    {
        from[index] = 0;
    }
}
template<typename T>
inline void truncate_buttom(T& from,size_t slices)
{
    TIPL_RUN(truncate_bottom_kernel,slices*from.plane_size())
            (tipl::make_shared(from),slices);
}

#include <hiprand/hiprand_kernel.h>

template<typename T>
__global__ void add_noise_kernel(T from,float noise_level)
{
    TIPL_FOR(index,from.size())
    {
        hiprandState state;
        hiprand_init(0, index, 0, &state);
        from[index] += noise_level*hiprand_uniform(&state);
    }
}
template<typename T>
inline void add_noise(T& from,float noise_level)
{
    TIPL_RUN(add_noise_kernel,from.size())
            (tipl::make_shared(from),noise_level);
}

template<typename T>
__global__ void diffuse_light_kernel(T from,tipl::vector<3> f,tipl::vector<3> center,float magnitude)
{
    TIPL_FOR(index,from.size())
    {
        from[index] *= std::max<float>(0.0f,1.0f + (tipl::vector<3>(tipl::pixel_index<3>(index,from.shape()))-center)*f);
    }
}

template<typename T>
inline void diffuse_light_cuda(T& from,tipl::vector<3> f,float magnitude)
{
    auto center = tipl::vector<3>(from.shape())*0.5f;
    f.normalize();
    f *= magnitude/float(tipl::max_value(from.shape().begin(),from.shape().end()));
    TIPL_RUN(diffuse_light_kernel,from.size())
            (tipl::make_shared(from),f,center,magnitude);
}


template<typename T>
__global__ void specular_light_kernel(T from,tipl::vector<3> center,float frequency,float mag,float b)
{
    TIPL_FOR(index,from.size())
    {
        from[index] *= ((std::cos((tipl::vector<3>(tipl::pixel_index<3>(index,from.shape()))-center).length()*frequency)+1.0f)*mag+b);
    }
}

template<typename T>
inline void specular_light_cuda(T& from,tipl::vector<3> center,float frequency,float mag)
{
    float b = 1.0f-mag-mag;
    frequency *= std::acos(-1)*0.5f/tipl::max_value(from.shape().begin(),from.shape().end());
    TIPL_RUN(specular_light_kernel,from.size())
            (tipl::make_shared(from),center,frequency,mag,b);
}
template<typename T>
__global__ void lens_distortion_kernel(T displaced,tipl::vector<3> center,float magnitude)
{
    TIPL_FOR(index,displaced.size())
    {
        tipl::vector<3> dir(tipl::pixel_index<3>(index,displaced.shape()));
        dir -= center;
        dir *= -magnitude*(dir.length2());
        displaced[index] = dir;
    }
}
template<typename image_type>
inline void lens_distortion_cuda(image_type& displaced,float magnitude)
{
    float radius = tipl::max_value(displaced.shape())/2;
    float radius2 = radius*radius;
    tipl::vector<3,int> center(displaced.shape());
    center /= 2;
    magnitude /= radius2;
    TIPL_RUN(lens_distortion_kernel,displaced.size())
            (tipl::make_shared(displaced),center,magnitude);
}

template<typename T>
__global__ void create_distortion_at_kernel(T displaced,tipl::vector<3> center,float radius,float radius_5,float pi_2_radius)
{
    TIPL_FOR(index,displaced.size())
    {
        tipl::vector<3> dir(tipl::pixel_index<3>(index,displaced.shape()));
        dir -= center;
        if(dir[0] > radius || dir[1] > radius || dir[2] > radius)
            return;
        auto length = dir.length();
        if(length > radius)
            return;
        dir *= -radius_5*std::sin(length*pi_2_radius)/length;
        displaced[index] += dir;
    }
}
template<typename image_type>
inline void create_distortion_at_cuda(image_type& displaced,const tipl::vector<3,int>& center,float radius,float magnitude)
{
    auto radius_5 = radius*magnitude;
    auto pi_2_radius = std::acos(-1)/radius;
    TIPL_RUN(create_distortion_at_kernel,displaced.size())
            (tipl::make_shared(displaced),center,radius,radius_5,pi_2_radius);
}

template<typename T>
__global__ void accumulate_transforms_kernel(T displaced,bool has_lens_distortion,bool has_perspective,
                                             tipl::vector<3> center,
                                             tipl::vector<3> perspective,
                                             tipl::transformation_matrix<float> trans)
{
    TIPL_FOR(index,displaced.size())
    {
        // pos now in the "retina" space
        tipl::vector<3> pos(tipl::pixel_index<3>(index,displaced.shape()));
        if(has_lens_distortion)
            pos += displaced[index];
        if(has_perspective)
            pos /= (perspective*(pos-center)+1.0f);
        // rigid motion + zoom + aspect ratio
        trans(pos);
        displaced[index] = pos;
    }
}




template<typename image_type>
inline void accumulate_transforms_cuda(image_type& displaced,bool has_lens_distortion,bool has_perspective,
                           const tipl::vector<3>& perspective,
                           const tipl::transformation_matrix<float>& trans)
{
    auto center = tipl::vector<3>(displaced.shape())/2.0f;
    TIPL_RUN(accumulate_transforms_kernel,displaced.size())
            (tipl::make_shared(displaced),has_lens_distortion,has_perspective,center,perspective,trans);
}


template<typename T1,typename T2,typename T3>
__global__ void blend_kernel(T1 image,T2 label,T3 background)
{
    TIPL_FOR(index,image.size())
    {
        if(!label[index])
            image[index] += background[index]*std::max<float>(0.1f,1.0f-image[index]);
    }
}



__INLINE__ float lerp_cuda(float t, float a, float b)	{return a + t * (b - a);}
__INLINE__ float fade_cuda(float t) 			{return t * t * t * (t * (t * 6.0f - 15.0f) + 10.0f);}
__INLINE__ float grad_cuda(int hash, float x, float y, float z) {
    int h = hash & 15;
    float u = h < 8 ? x : y;
    float v = h < 4 ? y : h == 12 || h == 14 ? x : z;
    return ((h & 1) ? -u : u) + ((h & 2) ? -v : v);
}

template<typename T>
__INLINE__ float perlin_texture_cuda(float x, float y, float z,T p)
{
    int xi = (int)floor(x) & 255;
    int yi = (int)floor(y) & 255;
    int zi = (int)floor(z) & 255;

    float xf = x - floor(x);
    float yf = y - floor(y);
    float zf = z - floor(z);

    float u = fade_cuda(xf);
    float v = fade_cuda(yf);
    float w = fade_cuda(zf);

    int aaa = p[p[p[xi] + yi] + zi];
    int aba = p[p[p[xi] + yi + 1] + zi];
    int aab = p[p[p[xi] + yi] + zi + 1];
    int abb = p[p[p[xi] + yi + 1] + zi + 1];
    int baa = p[p[p[xi + 1] + yi] + zi];
    int bba = p[p[p[xi + 1] + yi + 1] + zi];
    int bab = p[p[p[xi + 1] + yi] + zi + 1];
    int bbb = p[p[p[xi + 1] + yi + 1] + zi + 1];

    float x1 = lerp_cuda(u, grad_cuda(aaa, xf, yf, zf),
                        grad_cuda(baa, xf - 1, yf, zf));
    float x2 = lerp_cuda(u, grad_cuda(aba, xf, yf - 1, zf),
                        grad_cuda(bba, xf - 1, yf - 1, zf));
    float y1 = lerp_cuda(v, x1, x2);

    x1 = lerp_cuda(u, grad_cuda(aab, xf, yf, zf - 1),
                  grad_cuda(bab, xf - 1, yf, zf - 1));
    x2 = lerp_cuda(u, grad_cuda(abb, xf, yf - 1, zf - 1),
                  grad_cuda(bbb, xf - 1, yf - 1, zf - 1));
    float y2 = lerp_cuda(v, x1, x2);

    return lerp_cuda(w, y1, y2);
}

template<typename T,typename T2>
__global__ void perlin_texture_kernel(T background,T2 p,float scale,float pow_octave)
{
    TIPL_FOR(index,background.size())
    {
        tipl::vector<3> pos(tipl::pixel_index<3>(index,background.shape()));
        pos *= scale;
        background[index] += perlin_texture_cuda(pos[0],pos[1],pos[2],p.data())*pow_octave;
    }
}

template<typename T>
__global__ void perlin_texture_kernel2(T background)
{
    TIPL_FOR(index,background.size())
    {
        float v = background[index];
        v *= 2.0f;
        background[index] = v-std::floor(v);
    }
}



void visual_perception_augmentation_cuda(std::unordered_map<std::string,float>& options,
                          tipl::image<3>& input_,
                          tipl::image<3>& label_,
                          bool is_label,
                          const tipl::shape<3>& image_shape,
                          const tipl::vector<3>& image_vs,
                          size_t random_seed)
{
    try{

    {
        int gpu_count = 1;
        if(hipGetDeviceCount(&gpu_count) != hipSuccess ||
            hipSetDevice(gpu_count-1) != hipSuccess)
            tipl::out() << "hipSetDevice error:" << hipSetDevice(gpu_count-1) << std::endl;
    }

    tipl::device_image<3> input = input_;
    tipl::device_image<3> label = label_;

    tipl::uniform_dist<float> one(-1.0f,1.0f,random_seed);
    auto range = [&one](float from,float to){return one()*(to-from)*0.5f+(to+from)*0.5f;};
    auto apply = [&one,&options](const char* name)
    {
        int index = int(options[name]);
        if(index == 0)
            return false;
        if(index >= 4)
            return true;
        return std::abs(one()) < float(index)*0.25f;
    };

    auto random_location = [&range](const tipl::shape<3>& sp,float from,float to)
                    {return tipl::vector<3,int>((sp[0]-1)*range(from,to),(sp[1]-1)*range(from,to),(sp[2]-1)*range(from,to));};

    tipl::device_image<3> output(input.shape());
    std::vector<tipl::image<3,float,tipl::pointer_container> > input_images(input.depth()/image_shape[2]),output_images(input.depth()/image_shape[2]);
    for(size_t c = 0;c < input_images.size();++c)
    {
        input_images[c] = input.alias(c*image_shape.size(),image_shape);
        output_images[c] = output.alias(c*image_shape.size(),image_shape);
    }



    {
        bool downsample_x = apply("downsample_x");
        bool downsample_y = apply("downsample_y");
        bool downsample_z = apply("downsample_z");
        if(downsample_x || downsample_y || downsample_z)
        {
            tipl::device_image<3> low_reso_image(tipl::shape<3>(float(image_shape[0])*(downsample_x ? options["downsample_x_ratio"]: 1.0f),
                                                        float(image_shape[1])*(downsample_y ? options["downsample_y_ratio"]: 1.0f),
                                                        float(image_shape[2])*(downsample_z ? options["downsample_z_ratio"]: 1.0f)));
            for(auto& image : input_images)
            {
                tipl::scale_cuda(image,low_reso_image);
                tipl::scale_cuda(low_reso_image,image);
            }
        }
    }

    if(apply("cropping"))
    {
        auto cropping_size = range(options["cropping_size_min"],
                                   options["cropping_size_max"])*float(image_shape.width());
        auto cropping_value = range(0.0f,2.0f);
        auto location = random_location(image_shape,cropping_size,1.0f - cropping_size);
        for(auto& image : input_images)
            cropping_at(image,label,location,cropping_size,cropping_value);
    }

    if(apply("truncation_z"))
    {
        int num_top_slices = int(std::fabs(one()*0.5f*float(label.depth())));
        int num_bottom_slices = int(std::fabs(one()*0.5f*float(label.depth())));
        truncate_top(label,num_top_slices);
        truncate_buttom(label,num_bottom_slices);
        for(auto& image : input_images)
        {
            truncate_top(image,num_top_slices);
            truncate_buttom(image,num_bottom_slices);
        }
    }

    if(apply("noise"))
    {
        float noise_mag = options["noise_mag"];
        for(auto& image : input_images)
            add_noise(image,noise_mag);
    }
    // lighting
    if(apply("ambient"))
    {
        float ambient_magnitude = one()*options["ambient_mag"];
        for(auto& image : input_images)
            tipl::add_constant_cuda(image,ambient_magnitude);
    }
    if(apply("diffuse"))
    {
        auto diffuse_dir = tipl::vector<3>(one()-0.5f,one()-0.5f,one()-0.5f);
        for(auto& image : input_images)
            diffuse_light_cuda(image,diffuse_dir,options["diffuse_mag"]);
    }
    if(apply("specular"))
    {
        auto location = random_location(image_shape,0.4f,0.6f);
        for(auto& image : input_images)
            specular_light_cuda(image,location,options["specular_freq"],options["specular_mag"]);
    }

    // rigid motion + view port
    tipl::device_image<3> output_label(image_shape);
    {
        auto resolution = range(1.0f/options["scaling_up"],1.0f/options["scaling_down"]);
        tipl::affine_transform<float> transform = {
                    one()*float(options["translocation_ratio"])*image_shape[0]*image_vs[0],
                    one()*float(options["translocation_ratio"])*image_shape[1]*image_vs[1],
                    one()*float(options["translocation_ratio"])*image_shape[2]*image_vs[2],
                    one()*options["rotation_x"],
                    one()*options["rotation_y"],
                    one()*options["rotation_z"],
                    resolution*range(1.0f/options["aspect_ratio"],options["aspect_ratio"]),
                    resolution*range(1.0f/options["aspect_ratio"],options["aspect_ratio"]),
                    resolution*range(1.0f/options["aspect_ratio"],options["aspect_ratio"]),
                    0.0f,0.0f,0.0f};
        auto trans = tipl::transformation_matrix<float>(transform,image_shape,image_vs,image_shape,image_vs);


        tipl::vector<3> perspective((one()-0.5f)*options["perspective"]/float(image_shape[0]),
                                    (one()-0.5f)*options["perspective"]/float(image_shape[1]),
                                    (one()-0.5f)*options["perspective"]/float(image_shape[2]));
        auto center = tipl::vector<3>(image_shape)/2.0f;


        tipl::device_image<3,tipl::vector<3> > displaced(image_shape);
        if(options["lens_distortion"] != 0.0f)
            lens_distortion_cuda(displaced,one()*options["lens_distortion"]);

        if(apply("distortion"))
        {
            size_t num = size_t(range(1.0f,options["distortion_count"]+1.0f));
            for(size_t i = 0;i < num;++i)
                create_distortion_at_cuda(displaced,random_location(image_shape,0.3f,0.7f),
                                             float(image_shape[0])*range(
                                                options["distortion_radius_min"],
                                                options["distortion_radius_max"]), // radius
                                                range(
                                                options["distortion_mag_min"],
                                                options["distortion_mag_max"]));  //magnitude
        }


        accumulate_transforms_cuda(displaced,options["lens_distortion"] > 0.0f,options["perspective"] > 0.0f,perspective,trans);

        if(is_label)
            tipl::compose_mapping_cuda<tipl::nearest>(label,displaced,output_label);
        else
            tipl::compose_mapping_cuda(label,displaced,output_label);

        for(size_t c = 0;c < output_images.size();++c)
            tipl::compose_mapping_cuda(input_images[c],displaced,output_images[c]);
    }


    for(auto& image : output_images)
    {
        tipl::lower_threshold_cuda(image,0.0f);
        tipl::normalize_cuda(image);
    }

    // background
    if(!output_label.empty() && is_label)
    {

        if(apply("zero_background"))
        {
            for(auto& image : output_images)
                tipl::preserve_cuda(image,output_label);
            goto end;
        }


        if(apply("rubber_stamping"))
        {
            std::vector<tipl::affine_transform<float> > args;
            float pi2 = std::acos(-1)*2.0f;
            for(size_t iter = 0;iter < 5;++iter)
                args.push_back(tipl::affine_transform<float>{one()*image_shape[0]*image_vs[0]*0.5f,
                                                    one()*image_shape[1]*image_vs[1]*0.5f,
                                                    one()*image_shape[2]*image_vs[2]*0.5f,
                                                    one()*pi2,one()*pi2,one()*pi2,
                                                    range(0.8f,1.25f),range(0.8f,1.25f),range(0.8f,1.25f),
                                                    0.0f,0.0f,0.0f});
            for(int c = 0;c < input_images.size();++c)
            {
                auto& image = input_images[c];
                auto& image_out = output_images[c];

                tipl::masking_cuda(image,label);

                tipl::device_image<3> background(image_shape);
                for(size_t iter = 0;iter < 5;++iter)
                {
                    tipl::resample_cuda(image,background,tipl::transformation_matrix<float>(args[iter],image_shape,image_vs,image_shape,image_vs));
                    tipl::lower_threshold_cuda(background,0.0f);
                    tipl::normalize_cuda(background,options["rubber_stamping_mag"]);
                    TIPL_RUN(blend_kernel,image_out.size())
                            (tipl::make_shared(image_out),tipl::make_shared(output_label),tipl::make_shared(background));
                }
            }
        }

        if(apply("perlin_texture"))
        {
            std::vector<int> p(512);
            for(size_t i = 0;i < p.size();i++)
                p[i] = i & 255;
            std::shuffle(p.begin(), p.end(),std::mt19937(random_seed));

            tipl::device_vector<int> p_device = p;

            tipl::device_image<3> background(image_shape);
            float zoom = range(0.005f,0.05f);
            for (int octave = 0; octave < 4; octave++)
            {
                float pow_octave = pow(0.5f, octave);
                float scale = zoom * pow_octave;
                TIPL_RUN(perlin_texture_kernel,background.size())
                        (tipl::make_shared(background),tipl::make_shared(p_device),scale,pow_octave);
            }

            TIPL_RUN(perlin_texture_kernel2,background.size())
                    (tipl::make_shared(background));

            tipl::normalize_cuda(background,options["perlin_texture_mag"]);
            for(auto& image : output_images)
                TIPL_RUN(blend_kernel,image.size())
                (tipl::make_shared(image),tipl::make_shared(output_label),tipl::make_shared(background));

        }

        for(auto& image : output_images)
        {
            tipl::lower_threshold_cuda(image,0.0f);
            tipl::normalize_cuda(image);
        }

    }

    end:

    output.buf().copy_to(input_.buf());
    output_label.buf().copy_to(label_.buf());

    }
    catch(std::runtime_error& error)
    {
        tipl::out() << "ERROR: " << error.what() << std::endl;
    }
    if(hipSetDevice(0) != hipSuccess)
        tipl::out() << "hipSetDevice error:" << hipSetDevice(0) << std::endl;
}
